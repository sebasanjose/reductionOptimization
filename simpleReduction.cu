#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256

// Simple parallel reduction kernel using shared memory.
__global__ void simpleReductionKernel(int *input, int *output, int N) {
    __shared__ int sharedMem[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    
    // Load data from global memory into shared memory.
    sharedMem[tid] = (i < N) ? input[i] : 0;
    __syncthreads();

    // Perform reduction in shared memory.
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if(tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }
    
    // Write the block's sum to global memory.
    if(tid == 0) {
        output[blockIdx.x] = sharedMem[0];
    }
}

int main(){
    int N = 1 << 20; // 1 Million elements
    int size = N * sizeof(int);
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    
    int *h_input = (int*) malloc(size);
    int *h_output = (int*) malloc(numBlocks * sizeof(int));

    // Initialize input array (all ones).
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;
    }

    int *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, numBlocks * sizeof(int));

    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Create CUDA events for timing.
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Launch kernel for block-level reduction.
    simpleReductionKernel<<<numBlocks, BLOCK_SIZE>>>(d_input, d_output, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time elapsed for simpleReduction kernel: %f ms\n", elapsedTime);

    // Copy partial results back to host and perform final reduction on CPU.
    hipMemcpy(h_output, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
    int total = 0;
    for (int i = 0; i < numBlocks; i++){
        total += h_output[i];
    }
    printf("Total Sum: %d\n", total);

    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_output);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
