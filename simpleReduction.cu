#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define BLOCK_SIZE 256

// Kernel: simple reduction within a block using shared memory.
__global__ void simpleReductionKernel(int *input, int *output, int N) {
    __shared__ int sharedMem[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    sharedMem[tid] = (i < N) ? input[i] : 0;
    __syncthreads();
    
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = sharedMem[0];
    }
}

int main(){
    int N = 1 << 20; // 1,048,576 elements
    int size = N * sizeof(int);
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Allocate and initialize host memory.
    int *h_input = (int*) malloc(size);
    int *h_partialSums = (int*) malloc(numBlocks * sizeof(int));
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;
    }

    // Allocate device memory.
    int *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, numBlocks * sizeof(int));
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Create overall timing events.
    hipEvent_t start_total, stop_total;
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventRecord(start_total);

    // Launch the kernel.
    simpleReductionKernel<<<numBlocks, BLOCK_SIZE>>>(d_input, d_output, N);

    // Copy block-level partial sums back and finish the reduction on CPU.
    hipMemcpy(h_partialSums, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
    int total = 0;
    for (int i = 0; i < numBlocks; i++){
        total += h_partialSums[i];
    }

    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);
    float elapsedTime_total;
    hipEventElapsedTime(&elapsedTime_total, start_total, stop_total);
    printf("Total elapsed time for simpleReduction: %f ms\n", elapsedTime_total);
    printf("Total Sum: %d (expected %d)\n", total, N);

    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_partialSums);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
    return 0;
}
