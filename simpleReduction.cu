#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define BLOCK_SIZE 256
// Increase dataset size.
#define N (1 << 26)  // 67,108,864 elements

// Kernel: simple reduction within a block using shared memory.
__global__ void simpleReductionKernel(int *input, int *output, int n) {
    __shared__ int sharedMem[BLOCK_SIZE];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + tid;
    sharedMem[tid] = (i < n) ? input[i] : 0;
    __syncthreads();
    
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }
    
    if (tid == 0) {
        output[blockIdx.x] = sharedMem[0];
    }
}

int main(){
    int size = N * sizeof(int);
    int numBlocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;

    // Allocate and initialize host memory.
    int *h_input = (int*) malloc(size);
    int *h_partialSums = (int*) malloc(numBlocks * sizeof(int));
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;
    }

    // Allocate device memory.
    int *d_input, *d_output;
    hipMalloc(&d_input, size);
    hipMalloc(&d_output, numBlocks * sizeof(int));
    hipMemcpy(d_input, h_input, size, hipMemcpyHostToDevice);

    // Create overall timing events.
    hipEvent_t start_total, stop_total;
    hipEventCreate(&start_total);
    hipEventCreate(&stop_total);
    hipEventRecord(start_total);

    // Launch the kernel.
    simpleReductionKernel<<<numBlocks, BLOCK_SIZE>>>(d_input, d_output, N);

    // Copy partial sums back and finish the reduction on CPU.
    hipMemcpy(h_partialSums, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
    long long total = 0;
    for (int i = 0; i < numBlocks; i++){
        total += h_partialSums[i];
    }

    hipEventRecord(stop_total);
    hipEventSynchronize(stop_total);
    float elapsedTime_total;
    hipEventElapsedTime(&elapsedTime_total, start_total, stop_total);
    printf("Total elapsed time for simpleReduction: %f ms\n", elapsedTime_total);
    printf("Total Sum: %lld (expected %d)\n", total, N);

    // Cleanup.
    hipFree(d_input);
    hipFree(d_output);
    free(h_input);
    free(h_partialSums);
    hipEventDestroy(start_total);
    hipEventDestroy(stop_total);
    return 0;
}
