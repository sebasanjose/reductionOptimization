#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 256  // Threads per block

// First kernel: Block-wise reduction
__global__ void sumReduction(int *input, int *output, int N) {
    __shared__ int sharedMem[BLOCK_SIZE];

    int tid = threadIdx.x;
    int index = blockIdx.x * blockDim.x + tid;

    // Load elements from global memory into shared memory.
    sharedMem[tid] = (index < N) ? input[index] : 0;
    __syncthreads();

    // Parallel reduction in shared memory.
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    // Write block sum to global memory.
    if (tid == 0) {
        output[blockIdx.x] = sharedMem[0];
    }
}

// Second kernel: Final sum of partial sums.
__global__ void finalReduction(int *input, int *output, int N) {
    __shared__ int sharedMem[BLOCK_SIZE];

    int tid = threadIdx.x;
    int index = tid;

    // Load partial sums into shared memory.
    sharedMem[tid] = (index < N) ? input[index] : 0;
    __syncthreads();

    // Perform reduction.
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedMem[tid] += sharedMem[tid + stride];
        }
        __syncthreads();
    }

    // Store final sum in global memory.
    if (tid == 0) {
        *output = sharedMem[0];
    }
}

int main() {
    int N = 1 << 20;  // 1 Million elements
    int blockSize = BLOCK_SIZE;
    int numBlocks = (N + blockSize - 1) / blockSize;

    int *h_input, *h_output;
    int *d_input, *d_partialSums, *d_finalSum;

    // Allocate host memory.
    h_input = (int*) malloc(N * sizeof(int));
    h_output = (int*) malloc(sizeof(int));

    // Initialize input array.
    for (int i = 0; i < N; i++) {
        h_input[i] = 1;  // All elements are 1.
    }

    // Allocate GPU memory.
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_partialSums, numBlocks * sizeof(int));
    hipMalloc(&d_finalSum, sizeof(int));

    // Copy data to GPU.
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // Create events for timing.
    hipEvent_t start, stop;
    float elapsedTime1, elapsedTime2;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Launch first kernel and time it.
    hipEventRecord(start);
    sumReduction<<<numBlocks, blockSize>>>(d_input, d_partialSums, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime1, start, stop);
    printf("Time elapsed for sumReduction kernel: %f ms\n", elapsedTime1);

    // Launch second kernel (using 1 block) and time it.
    hipEventRecord(start);
    finalReduction<<<1, blockSize>>>(d_partialSums, d_finalSum, numBlocks);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime2, start, stop);
    printf("Time elapsed for finalReduction kernel: %f ms\n", elapsedTime2);

    // Copy result back to host.
    hipMemcpy(h_output, d_finalSum, sizeof(int), hipMemcpyDeviceToHost);
    printf("Total Sum: %d\n", *h_output); // Should print 1 million

    // Free memory.
    hipFree(d_input);
    hipFree(d_partialSums);
    hipFree(d_finalSum);
    free(h_input);
    free(h_output);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
